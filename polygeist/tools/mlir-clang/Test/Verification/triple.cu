
#include <hip/hip_runtime.h>
// Copyright (C) Codeplay Software Limited

// RUN: cgeist --target aarch64-unknown-linux-gnu %s %stdinclude -S -o - |
// FileCheck %s -check-prefix=MLIR RUN: cgeist --target
// aarch64-unknown-linux-gnu %s %stdinclude -emit-llvm -S -o - | FileCheck %s
// -check-prefix=LLVM

// XFAIL: *

// MLIR:  llvm.target_triple = "aarch64-unknown-linux-gnu"
// LLVM:  target triple = "aarch64-unknown-linux-gnu"

int main() { return 0; }
